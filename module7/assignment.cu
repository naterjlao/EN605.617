//-----------------------------------------------------------------------------
/// @file assignment.cu
/// @author Nate Lao (nlao1@jh.edu)
/// @brief Module 6 Main Driver
//-----------------------------------------------------------------------------
#include <stdio.h>
#include <math.h>
#include "helpers.cuh"
#include "transform.cuh"

#define PRINT_RESULTS 1
typedef void (*KERNEL_FUNCTION)(float *, size_t);
__host__ void setup(
	const int totalThreads,
	const int numBlocks,
	const int blockSize,
	float *buffer);
__host__ float run_serial(
	const int totalThreads,
	const int numBlocks,
	const int
		blockSize,
	float *buffer,
	const size_t buffer_size,
	KERNEL_FUNCTION kernel_function,
	const size_t iterations);

//-----------------------------------------------------------------------------
/// @brief Main Driver
//-----------------------------------------------------------------------------
int main(int argc, char **argv)
{
	// read command line arguments
	int totalThreads = (1 << 20);
	int blockSize = 256;

	if (argc >= 2)
	{
		totalThreads = atoi(argv[1]);
	}
	if (argc >= 3)
	{
		blockSize = atoi(argv[2]);
	}

	int numBlocks = totalThreads / blockSize;

	// validate command line arguments
	if (totalThreads % blockSize != 0)
	{
		++numBlocks;
		totalThreads = numBlocks * blockSize;

		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}

	// Setup 2D matrix, note that each thread operates on a coordinate
	// A coordinate is comprised of two points and the buffer datastructure
	// is modeled in an float[2][n_points] array.
	// Access to the coordinate at idx should be as follows:
	// x <- coordinates[idx]
	// y <- coordinates[idx + n_points]
	const size_t BUFFER_SIZE = 2 * totalThreads * sizeof(float);
	float *coordinates = (float *)malloc(BUFFER_SIZE);

	// Allocate coordinates, note that this is called twice for the x and y elements
	setup(totalThreads, numBlocks, blockSize, &coordinates[0]);
	setup(totalThreads, numBlocks, blockSize, &coordinates[totalThreads]);

#if PRINT_RESULTS
	for (size_t i = 0; i < totalThreads; i++)
		printf("(%f, %f)\n",coordinates[i], coordinates[i+totalThreads]);
#endif

	// Execute serial and async time tests
	const size_t ITERATIONS = 10;
	printf("%f,", run_serial(totalThreads, numBlocks, blockSize, coordinates, BUFFER_SIZE, kernel_call_global, ITERATIONS));
	// printf("%f\n", run_test(totalThreads, numBlocks, blockSize, coordinates, BUFFER_SIZE, kernel_call_register));

	// Cleanup
	free(coordinates);
}

//-----------------------------------------------------------------------------
/// @brief Sets up the working buffer by generating random floats
/// @param totalThreads Length of the buffer
/// @param numBlocks Cuda Number of Blocks
/// @param blockSize Cuda Block Size
/// @param buffer pointer to local host buffer
/// @return None; buffer is modified
//-----------------------------------------------------------------------------
__host__ void setup(
	const int totalThreads,
	const int numBlocks,
	const int blockSize,
	float *buffer)
{
	hiprandState *r_state;
	hipMalloc(&r_state, totalThreads * sizeof(hiprandState));
	setup_random<<<numBlocks, blockSize>>>(r_state);

	float *dev_buffer;
	hipMalloc(&dev_buffer, totalThreads * sizeof(float));

	// Create random F32s
	populate_random_floats<<<numBlocks, blockSize>>>(dev_buffer, r_state);

	hipMemcpy(buffer, dev_buffer, totalThreads * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(r_state);
	hipFree(dev_buffer);
}

//-----------------------------------------------------------------------------
/// @brief Executes a CUDA kernel in a serial memory access.
/// @param totalThreads Number of CUDA Threads
/// @param numBlocks Number of CUDA Blocks
/// @param blockSize CUDA Block Size
/// @param buffer Pointer to host buffer
/// @param buffer_size Size of the host buffer in bytes
/// @param kernel_function Pointer to the KERNEL_FUNCTION to execute.
/// @param interations Number of times to iterate the kernel_function.
/// @return The duration of time to execute the kernel function
//-----------------------------------------------------------------------------
__host__ float run_serial(
	const int totalThreads,
	const int numBlocks,
	const int blockSize,
	float *buffer,
	const size_t buffer_size,
	KERNEL_FUNCTION kernel_function,
	const size_t iterations)
{
	// Setup stopwatch
	hipEvent_t start;
	hipEvent_t end;
	float timer;
	hipEventCreate(&start);
	hipEventCreate(&end);

	// Allocate device buffer
	float *dev_buffer;
	hipMalloc(&dev_buffer, buffer_size);
	hipMemcpy(dev_buffer, buffer, buffer_size, hipMemcpyHostToDevice);

	// Execute the kernel function
	hipEventRecord(start);
	kernel_function<<<numBlocks, blockSize>>>(dev_buffer, totalThreads);
	hipDeviceSynchronize();
	hipEventRecord(end);

#if PRINT_RESULTS
	float *result = (float *)malloc(buffer_size);
	hipMemcpy(result, dev_buffer, buffer_size, hipMemcpyDeviceToHost);
	for (size_t idx = 0; idx < totalThreads; idx++)
		printf("(%f,%f)\n", result[idx], result[idx + totalThreads]);
	free(result);
#endif
	hipFree(dev_buffer);
	hipEventElapsedTime(&timer, start, end);
	return timer;
}
