//-----------------------------------------------------------------------------
/// @file assignment.cu
/// @author Nate Lao (nlao1@jh.edu)
/// @brief Module 6 Main Driver
//-----------------------------------------------------------------------------
#include <stdio.h>
#include <chrono>
#include <math.h>
#include "helpers.cuh"
#include "transform.cuh"

#define PRINT_RESULTS 0
typedef void (*KERNEL_FUNCTION)(float *, size_t);
__host__ void setup(const int totalThreads, const int numBlocks, const int blockSize, float *buffer);
__host__ std::chrono::duration<int64_t, std::nano> run_test(const int totalThreads, const int numBlocks, const int blockSize, float *buffer, const size_t buffer_size, KERNEL_FUNCTION kernel_function);

//-----------------------------------------------------------------------------
/// @brief Main Driver
//-----------------------------------------------------------------------------
int main(int argc, char **argv)
{
	// read command line arguments
	int totalThreads = (1 << 20);
	int blockSize = 256;

	if (argc >= 2)
	{
		totalThreads = atoi(argv[1]);
	}
	if (argc >= 3)
	{
		blockSize = atoi(argv[2]);
	}

	int numBlocks = totalThreads / blockSize;

	// validate command line arguments
	if (totalThreads % blockSize != 0)
	{
		++numBlocks;
		totalThreads = numBlocks * blockSize;

		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}

	// Setup 2D matrix, note that each thread operates on a coordinate
	// A coordinate is comprised of two points and the buffer datastructure
	// is modeled in an float[2][n_points] array.
	// Access to the coordinate at idx should be as follows:
	// x <- coordinates[idx]
	// y <- coordinates[idx + n_points]
	const size_t BUFFER_SIZE = 2 * totalThreads * sizeof(float);
	float *coordinates = (float *)malloc(BUFFER_SIZE);

	// Allocate coordinates, note that this is called twice for the x and y elements
	setup(totalThreads, numBlocks, blockSize, &coordinates[0]);
	setup(totalThreads, numBlocks, blockSize, &coordinates[totalThreads]);

	// Execute register and global time tests
	printf("%d,", run_test(totalThreads, numBlocks, blockSize, coordinates, BUFFER_SIZE, kernel_call_global));
	printf("%d\n", run_test(totalThreads, numBlocks, blockSize, coordinates, BUFFER_SIZE, kernel_call_register));

	// Cleanup
	free(coordinates);
}

//-----------------------------------------------------------------------------
/// @brief Sets up the working buffer by generating random floats
/// @param totalThreads Length of the buffer
/// @param numBlocks Cuda Number of Blocks
/// @param blockSize Cuda Block Size
/// @param buffer pointer to local host buffer
/// @return None; buffer is modified
//-----------------------------------------------------------------------------
__host__ void setup(const int totalThreads, const int numBlocks, const int blockSize, float *buffer)
{
	hiprandState *r_state;
	hipMalloc(&r_state, totalThreads * sizeof(hiprandState));
	setup_random<<<numBlocks, blockSize>>>(r_state);

	float *dev_buffer;
	hipMalloc(&dev_buffer, totalThreads * sizeof(float));

	// Create random F32s
	populate_random_floats<<<numBlocks, blockSize>>>(dev_buffer, r_state);

	hipMemcpy(buffer, dev_buffer, totalThreads * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(r_state);
	hipFree(dev_buffer);
}

//-----------------------------------------------------------------------------
/// @brief Executes a CUDA kernel call timed test
/// @param totalThreads Number of CUDA Threads
/// @param numBlocks Number of CUDA Blocks
/// @param blockSize CUDA Block Size
/// @param buffer Pointer to host buffer
/// @param buffer_size Size of the host buffer in bytes
/// @param kernel_function Pointer to a KERNEL_FUNCTION
/// @return The duration of time to execute the kernel function
//-----------------------------------------------------------------------------
__host__ std::chrono::duration<int64_t, std::nano> run_test(
	const int totalThreads, const int numBlocks, const int blockSize,
	float *buffer, const size_t buffer_size, KERNEL_FUNCTION kernel_function)
{
#if PRINT_RESULTS
	float *result = (float *)malloc(buffer_size);
#endif
	// Allocate device buffer
	float *dev_buffer;
	hipMalloc(&dev_buffer, buffer_size);
	hipMemcpy(dev_buffer, buffer, buffer_size, hipMemcpyHostToDevice);

	// Execute the kernel function
	const std::chrono::time_point<std::chrono::steady_clock> start = std::chrono::steady_clock::now();
	kernel_function<<<numBlocks, blockSize>>>(dev_buffer, totalThreads);
	hipDeviceSynchronize();
	const std::chrono::time_point<std::chrono::steady_clock> end = std::chrono::steady_clock::now();

#if PRINT_RESULTS
	hipMemcpy(result, dev_buffer, buffer_size, hipMemcpyDeviceToHost);
	for (size_t idx = 0; idx < totalThreads; idx++)
		printf("(%f,%f)\n", result[idx], result[idx + totalThreads]);
	free(result);
#endif
	hipFree(dev_buffer);
	return end - start;
}
