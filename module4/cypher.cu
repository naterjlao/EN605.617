#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
/// @file cypher.cu
/// @author Nate Lao (nlao1@jh.edu)
/// @brief Module 4 Caesar Cypher Implementation
//-----------------------------------------------------------------------------
#include "cypher.h"

//-----------------------------------------------------------------------------
/// @brief Offsets the character buffer by a given value.
/// @param buffer Pointer to character buffer.
/// @param offset Offset value.
/// @return None.
//-----------------------------------------------------------------------------
__global__ void caesar_cypher(char *buffer, const int offset)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    buffer[thread_idx] = buffer[thread_idx] + offset;
}