#include <stdio.h>
#include "fileio.h"
#include "cypher.h"


int main(int argc, char** argv)
{
	// Evaluate arguments
	if (argc != 2)
	{
		fprintf(stderr,"ERROR: invalid arguments, expecting:\n");
		fprintf(stderr,"main <FILENAME>\n");
		return -1;
	}

	// Read file input
	const char *filename = argv[1];
	char *buffer;
	size_t length = read_file(filename, &buffer);
	if (length == 0)
	{
		fprintf(stderr,"ERROR: empty file or file open failed\n");
		return -1;
	}

	// Setup Device Memory
	const int BLOCK_SIZE = 256;
	const int NUM_BLOCKS = length / BLOCK_SIZE + ((length % BLOCK_SIZE > 0) ? 1 : 0);
	char *device_buffer;
	hipMalloc(&device_buffer, (NUM_BLOCKS * BLOCK_SIZE) * sizeof(char)); // We allocate more if needed
	hipMemcpy(device_buffer, buffer, length, hipMemcpyHostToDevice);

	// Run the conversion on the GPU
	caesar_cypher<<<NUM_BLOCKS, BLOCK_SIZE>>>(device_buffer, 5);

	// Free Device Memory
	hipMemcpy(buffer, device_buffer, length, hipMemcpyDeviceToHost);
	hipFree(device_buffer);
	device_buffer = 0;

	// Output the conversion to stdout
	printf("%s\n",buffer);

	// Free Host Memory
	if (buffer != 0)
	{
		free(buffer);
		buffer = 0;
	}

	return 0;
}
